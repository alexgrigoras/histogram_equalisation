#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <assert.h>
#include <cstdlib>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <math.h>
#include <string.h>
#include <iostream>

using namespace cv;
using namespace std;

void print_array(int* vect, int  dim)
{
    for (long i = 0; i < dim; i++) printf("%d ", vect[i]);
}

void print_array(float* vect, int  dim)
{
    for (long i = 0; i < dim; i++) printf("%f ", vect[i]);
}

void display_histogram(int histogram[], const char* name) {
    int histogramWidth = 512;
    int histogramHeight = 400;
    int newHistogram[256];
    int binWidth;
    int maximumIntensity;

    for (int i = 0; i < 256; i++) newHistogram[i] = histogram[i];

    //creating "bins" for the range of 256 intensity values
    binWidth = cvRound((double)histogramWidth / 256);
    Mat histogramImage(histogramHeight, histogramWidth, CV_8UC1, Scalar(255, 255, 255));

    //finding maximum intensity level in the histogram
    maximumIntensity = newHistogram[0];
    for (int i = 1; i < 256; i++) {
        if (maximumIntensity < newHistogram[i]) maximumIntensity = newHistogram[i];
    }

    //normalizing histogram in terms of rows (y)
    for (int i = 0; i < 256; i++) newHistogram[i] = ((double)newHistogram[i] / maximumIntensity) * histogramImage.rows;

    //drawing the intensity level - line
    for (int i = 0; i < 256; i++) line(histogramImage, Point(binWidth * (i), histogramHeight), Point(binWidth * (i), histogramHeight - newHistogram[i]), Scalar(0, 0, 0), 1, 8, 0);
    
    namedWindow(name, WINDOW_AUTOSIZE);
    imshow(name, histogramImage);
}

__global__ void histogramKernel(int* d_out, int* d_in) {
    int in = blockIdx.x * blockDim.x + threadIdx.x;
    int value = d_in[in];

    atomicAdd(&d_out[value], 1);
}

__global__ void cumHistKernel(int* d_out, int* d_in)
{
    int in = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int cdf_val = 0;

    for (int i = 0; i < in; ++i)
    {
        cdf_val = cdf_val + d_in[i];
    }

    d_out[in] = cdf_val;
}

__global__ void prkKernel(float* d_out, int* d_in, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = (float)d_in[i] / size;
}

__global__ void skKernel(int* d_out, int* d_in, float alpha)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = round((float)d_in[i] * alpha);
}

__global__ void pskKernel(float* d_out, int* d_in_a, float* d_in_b)
{
    int in = blockIdx.x * blockDim.x + threadIdx.x;
    int out = (int)d_in_a[in];

    atomicAdd(&d_out[out], d_in_b[in]);
}

__global__ void finalValuesKernel(int* d_out, float* d_in)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = round(d_in[i] * 255);
}

__global__ void finalImageKernel(int* d_out, int* d_in, int* d_img)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = (uchar)(d_in[d_img[i]]);
}

int main()
{
    /*
    string image_str = "../images/img0";
    string extension = ".jpg";
    string img_name = image_str + extension;
    */
    Mat image = imread("D:/University/Master/Year 2/GPUP/Project/histogram_equalization/hist_equ_cuda/x64/Debug/img0.jpg", IMREAD_GRAYSCALE);

    int h = image.rows, w = image.cols;                             // image dimensions
    int *h_hist, *d_hist;
    int *h_image, *d_image;
    float *d_PRk;
    int *d_cumHist;
    int *d_Sk;
    float *d_PSk;
    int *h_finalValues, *d_finalValues;
    int* d_finalImage;
    int dim_hist = 256;
    int dim_image = h * w;                                          // image size
    float alpha = 255.0 / dim_image;
    hipError_t cudaStatus;
    int numThreadsPerBlock = 256;                                   // define block size
    int numBlocks = dim_image / numThreadsPerBlock;
    //hipEvent_t start, stop;
    float elapsedTime;

    //hipEventCreate(&start);
    //hipEventCreate(&stop);

    h_hist = new int[dim_hist];
    h_image = new int[dim_image];
    h_finalValues = new int[dim_hist];

    for (int i = 0; i < dim_hist; ++i) h_hist[i] = 0;

    for (int i = 0; i < h; i++) {
        for (int j = 0; j < w; j++) {
            h_image[i * w + j] = image.at<uchar>(i, j);
        }
    }

    // Check CUDA device
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    //hipEventRecord(start, 0);  // Start global timers

    // ******************************************************************************************
    // Compute image histogram

    // Copy host array to device array
    cudaStatus = hipMalloc((void**)&d_image, dim_image * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&d_hist, dim_hist * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_hist, h_hist, dim_hist * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_image, h_image, dim_image * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // launch kernel
    histogramKernel <<< numBlocks, numThreadsPerBlock >>> (d_hist, d_image);
    
    // block until the device has completed
    hipDeviceSynchronize();
    // device to host copy
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(h_hist, d_hist, dim_hist * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //display_histogram(h_hist, "CUDA Histogram");

    // ******************************************************************************************
    // Compute Cumulative Histogram 

    cudaStatus = hipMalloc((void**)&d_cumHist, dim_hist * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cumHistKernel <<< 1, dim_hist >> > (d_cumHist, d_hist);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // ******************************************************************************************
    // Probability distribution for intensity levels

    cudaStatus = hipMalloc((void**)&d_PRk, dim_hist * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    prkKernel <<< 1, dim_hist >>> (d_PRk, d_hist, dim_image);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // ******************************************************************************************
    // Scaling operation

    cudaStatus = hipMalloc((void**)&d_Sk, dim_hist * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    skKernel <<< 1, dim_hist >>> (d_Sk, d_cumHist, alpha);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // ******************************************************************************************
    // Mapping operation

    cudaStatus = hipMalloc((void**)&d_PSk, dim_hist * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    pskKernel << < 1, dim_hist >> > (d_PSk, d_Sk, d_PRk);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // ******************************************************************************************
    // Rounding to get final values

    cudaStatus = hipMalloc((void**)&d_finalValues, dim_hist * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    finalValuesKernel << < 1, dim_hist >> > (d_finalValues, d_PSk);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    cudaStatus = hipMemcpy(h_finalValues, d_finalValues, dim_hist * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    //display_histogram(h_finalValues, "CUDA Equalized histogram");

    // ******************************************************************************************
    // Creating equalized image

    cudaStatus = hipMalloc((void**)&d_finalImage, dim_image * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    finalImageKernel <<< numBlocks, numThreadsPerBlock >>> (d_finalImage, d_Sk, d_image);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    cudaStatus = hipMemcpy(h_image, d_finalImage, dim_image * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    for (int i = 0; i < h; i++) {
        for (int j = 0; j < w; j++) {
            image.at<uchar>(i, j) = h_image[i * w + j];
        }
    }

    /*
    hipEventRecord(stop, 0);                           
    hipEventSynchronize(stop);                         
    hipEventElapsedTime(&elapsedTime, start, stop);    // hipEventElapsedTime returns value in milliseconds.Resolution ~0.5ms
    printf("Execution time GPU: %f\n", elapsedTime);
    */

Error:
    // Free device memory
    hipFree(d_hist);
    hipFree(d_image);
    hipFree(d_PRk);
    hipFree(d_cumHist);
    hipFree(d_Sk);
    hipFree(d_PSk);
    hipFree(d_finalValues);
    hipFree(d_finalImage);
    // Free host memory
    std::free(h_hist);
    std::free(h_image);
    std::free(h_finalValues);
    // Destroy CUDA Event API Events
    //hipEventDestroy(start);
    //hipEventDestroy(stop);

    // Display equalized image
    /*
    namedWindow("CUDA Equilized Image", WINDOW_NORMAL);
    imshow("CUDA Equilized Image", image);
    waitKey();
    */

    return 0;
}