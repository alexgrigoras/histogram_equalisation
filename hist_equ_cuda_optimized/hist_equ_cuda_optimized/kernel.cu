#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <assert.h>
#include <cstdlib>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <math.h>
#include <string.h>
#include <iostream>

using namespace cv;
using namespace std;

#define BLOCK_SIZE 1024
#define PRIVATE 1024

void print_array(int* vect, int  dim)
{
    for (long i = 0; i < dim; i++) printf("%d ", vect[i]);
}

void print_array(float* vect, int  dim)
{
    for (long i = 0; i < dim; i++) printf("%f ", vect[i]);
}

void display_histogram(int histogram[], const char* name) {
    int histogramWidth = 512;
    int histogramHeight = 400;
    int newHistogram[256];
    int binWidth;
    int maximumIntensity;

    for (int i = 0; i < 256; i++) newHistogram[i] = histogram[i];

    //creating "bins" for the range of 256 intensity values
    binWidth = cvRound((double)histogramWidth / 256);
    Mat histogramImage(histogramHeight, histogramWidth, CV_8UC1, Scalar(255, 255, 255));

    //finding maximum intensity level in the histogram
    maximumIntensity = newHistogram[0];
    for (int i = 1; i < 256; i++) {
        if (maximumIntensity < newHistogram[i]) maximumIntensity = newHistogram[i];
    }

    //normalizing histogram in terms of rows (y)
    for (int i = 0; i < 256; i++) newHistogram[i] = ((double)newHistogram[i] / maximumIntensity) * histogramImage.rows;

    //drawing the intensity level - line
    for (int i = 0; i < 256; i++) line(histogramImage, Point(binWidth * (i), histogramHeight), Point(binWidth * (i), histogramHeight - newHistogram[i]), Scalar(0, 0, 0), 1, 8, 0);

    namedWindow(name, WINDOW_AUTOSIZE);
    imshow(name, histogramImage);
}

__global__ void histogramKernel(int* bins, long* input, long numElems) {
    int tx = threadIdx.x; int bx = blockIdx.x;

    // compute global thread coordinates
    int i = (bx * blockDim.x) + tx;

    // create a private histogram copy for each thread block
    __shared__ unsigned int hist[PRIVATE];

    // each thread must initialize more than 1 location
    if (PRIVATE > BLOCK_SIZE) {
        for (int j = tx; j < PRIVATE; j += BLOCK_SIZE) {
            if (j < PRIVATE) {
                hist[j] = 0;
            }
        }
    }
    // use the first `PRIVATE` threads of each block to init
    else {
        if (tx < PRIVATE) {
            hist[tx] = 0;
        }
    }
    // wait for all threads in the block to finish
    __syncthreads();

    // update private histogram
    if (i < numElems) {
        atomicAdd(&(hist[input[i]]), 1);
    }
    // wait for all threads in the block to finish
    __syncthreads();

    // each thread must update more than 1 location
    if (PRIVATE > BLOCK_SIZE) {
        for (int j = tx; j < PRIVATE; j += BLOCK_SIZE) {
            if (j < PRIVATE) {
                atomicAdd(&(bins[j]), hist[j]);
            }
        }
    }
    // use the first `PRIVATE` threads to update final histogram
    else {
        if (tx < PRIVATE) {
            atomicAdd(&(bins[tx]), hist[tx]);
        }
    }
}

// Shared memory using balanced trees (optimization)
__global__ void cumHistKernelBT(int* g_odata, int* g_idata, int n)
{
    extern __shared__ int temp[]; // allocated on invocation
    int thid = threadIdx.x;
    int offset = 1;
    temp[2 * thid] = g_idata[2 * thid]; // load input into shared memory
    temp[2 * thid + 1] = g_idata[2 * thid + 1];

    for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (thid < d)
        {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;

            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (thid == 0) { temp[n - 1] = 0; } // clear the last element

    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();

        if (thid < d)
        {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    __syncthreads();
    g_odata[2 * thid] = temp[2 * thid]; // write results to device memory
    g_odata[2 * thid + 1] = temp[2 * thid + 1];
}

__global__ void prkKernel(float* d_out, int* d_in, long size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = (float)d_in[i] / size;
}

__global__ void skKernel(int* d_out, int* d_in, float alpha)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = round((float)d_in[i] * alpha);
}

__global__ void pskKernel(float* d_out, int* d_in_a, float* d_in_b)
{
    int in = blockIdx.x * blockDim.x + threadIdx.x;
    int out = (int)d_in_a[in];

    atomicAdd(&d_out[out], d_in_b[in]);
}

__global__ void finalValuesKernel(int* d_out, float* d_in)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = round(d_in[i] * 255);
}

__global__ void finalImageKernel(long* d_out, int* d_in)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = (uchar)(d_in[d_out[i]]);
}

int main()
{
    char img_path[1024];

    printf("Starting application\n");
    printf("Insert image path: ");

    scanf_s("%1023[^\n]", img_path, (unsigned)_countof(img_path));

    printf("Showing results\n");

    Mat image = imread(img_path, IMREAD_GRAYSCALE);
    int h = image.rows, w = image.cols;                             // image dimensions
    int* h_hist;
    long* h_image;
    float* h_PRk;
    int* h_cumHist;
    int* h_Sk;
    float* h_PSk;
    int* h_finalValues;
    int dim_hist = 256;
    long dim_image = h * w;                                          // image size
    float alpha = 255.0 / dim_image;
    hipError_t cudaStatus;
    int numThreadsPerBlock = 256;                                   // define block size
    int numBlocks = dim_image / numThreadsPerBlock;
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);  // Start global timers

    hipMallocManaged(&h_hist, dim_hist * sizeof(int));
    hipMallocManaged(&h_image, dim_image * sizeof(long));
    hipMallocManaged(&h_PRk, dim_hist * sizeof(float));
    hipMallocManaged(&h_cumHist, dim_hist * sizeof(int));
    hipMallocManaged(&h_Sk, dim_hist * sizeof(int));
    hipMallocManaged(&h_PSk, dim_hist * sizeof(float));
    hipMallocManaged(&h_finalValues, dim_hist * sizeof(int));

    for (int i = 0; i < dim_hist; ++i) h_hist[i] = 0;

    for (int i = 0; i < h; i++) {
        for (int j = 0; j < w; j++) {
            h_image[i * w + j] = image.at<uchar>(i, j);
        }
    }

    // Check CUDA device
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // ******************************************************************************************
    // Compute image histogram

    // launch kernel
    dim3 threadPerBlock(BLOCK_SIZE, 1, 1);
    dim3 blockPerGrid(ceil(dim_image / (float)BLOCK_SIZE), 1, 1);
    histogramKernel << <blockPerGrid, threadPerBlock >> > (h_hist, h_image, dim_image);

    // block until the device has completed
    hipDeviceSynchronize();
    // device to host copy
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel histo launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    display_histogram(h_hist, "CUDA Histogram");

    // ******************************************************************************************
    // Compute Cumulative Histogram 

    cumHistKernelBT << < 1, dim_hist, 2 * dim_hist * sizeof(int) >> > (h_cumHist, h_hist, dim_hist);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "[cumhist] addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // ******************************************************************************************
    // Probability distribution for intensity levels
    prkKernel << < 1, dim_hist >> > (h_PRk, h_hist, dim_image);

    // Scaling operation
    skKernel << < 1, dim_hist >> > (h_Sk, h_cumHist, alpha);

    // Mapping operation
    pskKernel << < 1, dim_hist >> > (h_PSk, h_Sk, h_PRk);

    hipDeviceSynchronize();

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    // ******************************************************************************************
    // Rounding to get final values
    finalValuesKernel << < 1, dim_hist >> > (h_finalValues, h_PSk);

    // Creating equalized image
    finalImageKernel << < numBlocks, numThreadsPerBlock >> > (h_image, h_Sk);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // ******************************************************************************************

    display_histogram(h_finalValues, "CUDA Equalized histogram");

    for (int i = 0; i < h; i++) {
        for (int j = 0; j < w; j++) {
            image.at<uchar>(i, j) = h_image[i * w + j];
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);    // hipEventElapsedTime returns value in milliseconds.Resolution ~0.5ms
    //printf("Execution time GPU: %f\n", elapsedTime);
    printf("%f\n", elapsedTime);

Error:
    // Free device memory
    hipFree(h_hist);
    hipFree(h_image);
    hipFree(h_PRk);
    hipFree(h_cumHist);
    hipFree(h_Sk);
    hipFree(h_PSk);
    hipFree(h_finalValues);
    // Free host memory
    // Destroy CUDA Event API Events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Display equalized image
    namedWindow("CUDA Equilized Image", WINDOW_NORMAL);
    imshow("CUDA Equilized Image", image);
    waitKey();

    return 0;
}