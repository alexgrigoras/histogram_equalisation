#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <assert.h>
#include <cstdlib>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <math.h>
#include <string.h>
#include <iostream>

using namespace cv;
using namespace std;

void print_array(int* vect, int  dim)
{
    for (long i = 0; i < dim; i++) printf("%d ", vect[i]);
}

void print_array(float* vect, int  dim)
{
    for (long i = 0; i < dim; i++) printf("%f ", vect[i]);
}

void display_histogram(int histogram[], const char* name) {
    int histogramWidth = 512;
    int histogramHeight = 400;
    int newHistogram[256];
    int binWidth;
    int maximumIntensity;

    for (int i = 0; i < 256; i++) newHistogram[i] = histogram[i];

    //creating "bins" for the range of 256 intensity values
    binWidth = cvRound((double)histogramWidth / 256);
    Mat histogramImage(histogramHeight, histogramWidth, CV_8UC1, Scalar(255, 255, 255));

    //finding maximum intensity level in the histogram
    maximumIntensity = newHistogram[0];
    for (int i = 1; i < 256; i++) {
        if (maximumIntensity < newHistogram[i]) maximumIntensity = newHistogram[i];
    }

    //normalizing histogram in terms of rows (y)
    for (int i = 0; i < 256; i++) newHistogram[i] = ((double)newHistogram[i] / maximumIntensity) * histogramImage.rows;

    //drawing the intensity level - line
    for (int i = 0; i < 256; i++) line(histogramImage, Point(binWidth * (i), histogramHeight), Point(binWidth * (i), histogramHeight - newHistogram[i]), Scalar(0, 0, 0), 1, 8, 0);

    namedWindow(name, WINDOW_AUTOSIZE);
    imshow(name, histogramImage);
}

// Hillis & Steele Parallel Scan Algorithm
__global__ void cumHistKernelHS(int* d_out, int* d_in, int n)
{
    int idx = threadIdx.x;
    extern __shared__ int temp[];
    int pout = 0, pin = 1;

    temp[idx] = (idx > 0) ? d_in[idx - 1] : 0;
    __syncthreads();

    for (int offset = 1; offset < n; offset *= 2) {
        // swap double buffer indices
        pout = 1 - pout;
        pin = 1 - pout;
        if (idx >= offset) {
            temp[pout * n + idx] = temp[pin * n + idx - offset] + temp[pin * n + idx];  // changed line
        }
        else {
            temp[pout * n + idx] = temp[pin * n + idx];
        }
        __syncthreads();
    }
    d_out[idx] = temp[pout * n + idx];
}

// Shared memory using balanced trees (optimization)
__global__ void cumHistKernelBT(int* g_odata, int* g_idata, int n)
{
    extern __shared__ int temp[]; // allocated on invocation
    int thid = threadIdx.x;
    int offset = 1;
    temp[2 * thid] = g_idata[2 * thid]; // load input into shared memory
    temp[2 * thid + 1] = g_idata[2 * thid + 1];

    for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
    {
        __syncthreads();
        if (thid < d)
        {
        int ai = offset * (2 * thid + 1) - 1;
        int bi = offset * (2 * thid + 2) - 1;
            
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (thid == 0) { temp[n - 1] = 0; } // clear the last element

    for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();

            if (thid < d)
            {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
                int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
            }
    }
    __syncthreads();
    g_odata[2 * thid] = temp[2 * thid]; // write results to device memory
    g_odata[2 * thid + 1] = temp[2 * thid + 1];
}

__global__ void histogramKernel(int* d_out, int* d_in, long size)
{
    extern __shared__ unsigned int tempHist[];
    int tx = threadIdx.x;
    unsigned int idx = tx + blockIdx.x * blockDim.x;

    tempHist[tx] = 0;
    __syncthreads();
    if (idx < size) {
        atomicAdd(&(tempHist[d_in[idx]]), 1);       // add to private histogram
    }
    __syncthreads();
    atomicAdd(&(d_out[tx]), tempHist[tx]);          // contribute to global histogram.
}

__global__ void prkKernel(float* d_out, int* d_in, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = (float)d_in[i] / size;
}

__global__ void skKernel(int* d_out, int* d_in, float alpha)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = round((float)d_in[i] * alpha);
}

__global__ void pskKernel(float* d_out, int* d_in_a, float* d_in_b)
{
    int in = blockIdx.x * blockDim.x + threadIdx.x;
    int out = (int)d_in_a[in];

    atomicAdd(&d_out[out], d_in_b[in]);
}

__global__ void finalValuesKernel(int* d_out, float* d_in)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = round(d_in[i] * 255);
}

__global__ void finalImageKernel(int* d_out, int* d_in)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    d_out[i] = (uchar)(d_in[d_out[i]]);
}

int main()
{
    /*
    string image_str = "../images/img0";
    string extension = ".jpg";
    string img_name = image_str + extension;
    */
    Mat image = imread("D:/University/Master/Year 2/GPUP/Project/histogram_equalization/hist_equ_cuda/x64/Debug/img0.jpg", IMREAD_GRAYSCALE);

    int h = image.rows, w = image.cols;                             // image dimensions
    int* h_hist;
    int* h_image;
    float* h_PRk;
    int* h_cumHist;
    int* h_Sk;
    float* h_PSk;
    int* h_finalValues;
    int dim_hist = 256;
    int dim_image = h * w;                                          // image size
    float alpha = 255.0 / dim_image;
    hipError_t cudaStatus;
    int numThreadsPerBlock = 256;                                   // define block size
    int numBlocks = dim_image / numThreadsPerBlock;
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMallocManaged(&h_hist, dim_hist * sizeof(int));
    hipMallocManaged(&h_image, dim_image * sizeof(int));
    hipMallocManaged(&h_PRk, dim_hist * sizeof(float));
    hipMallocManaged(&h_cumHist, dim_hist * sizeof(int));
    hipMallocManaged(&h_Sk, dim_hist * sizeof(int));
    hipMallocManaged(&h_PSk, dim_hist * sizeof(float));
    hipMallocManaged(&h_finalValues, dim_hist * sizeof(int));

    for (int i = 0; i < dim_hist; ++i) h_hist[i] = 0;

    for (int i = 0; i < h; i++) {
        for (int j = 0; j < w; j++) {
            h_image[i * w + j] = image.at<uchar>(i, j);
        }
    }

    // Check CUDA device
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipEventRecord(start, 0);  // Start global timers

    // ******************************************************************************************
    // Compute image histogram

    // launch kernel
    histogramKernel << < numBlocks, numThreadsPerBlock, dim_hist * sizeof(int) >> > (h_hist, h_image, dim_image);
    
    // block until the device has completed
    hipDeviceSynchronize();
    // device to host copy
    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    display_histogram(h_hist, "CUDA Histogram");

    // ******************************************************************************************
    // Compute Cumulative Histogram 

    cumHistKernelHS << < 1, dim_hist, 2 * dim_hist * sizeof(int) >> > (h_cumHist, h_hist, dim_hist);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "[cumhist] addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // ******************************************************************************************
    // Probability distribution for intensity levels

    prkKernel << < 1, dim_hist >> > (h_PRk, h_hist, dim_image);
    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // ******************************************************************************************
    // Scaling operation

    skKernel << < 1, dim_hist >> > (h_Sk, h_cumHist, alpha);
    hipDeviceSynchronize();

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
 
    // ******************************************************************************************
    // Mapping operation

    for (int i = 0; i < 256; i++) h_PSk[i] = 0.0;
   
    pskKernel << < 1, dim_hist >> > (h_PSk, h_Sk, h_PRk);
    
    hipDeviceSynchronize();    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    
    // ******************************************************************************************
    // Rounding to get final values

    finalValuesKernel <<< 1, dim_hist >>> (h_finalValues, h_PSk);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    display_histogram(h_finalValues, "CUDA Equalized histogram");

    // ******************************************************************************************
    // Creating equalized image
    
    finalImageKernel << < numBlocks, numThreadsPerBlock >> > (h_image, h_Sk);

    hipDeviceSynchronize();
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "[final] addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }
    
    for (int i = 0; i < h; i++) {
        for (int j = 0; j < w; j++) {
            image.at<uchar>(i, j) = h_image[i * w + j];
        }
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);    // hipEventElapsedTime returns value in milliseconds.Resolution ~0.5ms
    printf("Execution time GPU: %f\n", elapsedTime);

Error:
    // Free device memory
    hipFree(h_hist);
    hipFree(h_image);
    hipFree(h_PRk);
    hipFree(h_cumHist);
    hipFree(h_Sk);
    hipFree(h_PSk);
    hipFree(h_finalValues);
    // Free host memory
    // Destroy CUDA Event API Events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Display equalized image
    namedWindow("CUDA Equilized Image", WINDOW_NORMAL);
    imshow("CUDA Equilized Image", image);

    waitKey();

    return 0;
}